#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include "Bitmap.h"

#include <cstdint>
#include <math.h>
#include <string>
#include <iostream>

// Kernel executed at the Device
__global__ void fractalKernel(uint8_t* dev_a, const unsigned width, const unsigned height, const unsigned iterations)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;

    if ((x + y * width) < width * height)
    {
        double cr = x * 0.02 / width - 0.65;
        double ci = y * 0.02 / height - 0.4;
        double zr = 0.0, zi = 0.0;
        double re = 0.0, im = 0.0;

        int n = 0;
        while ((zr * zr + zi * zi) < 4.0 && n < iterations)
        {
            re = zr * zr - zi * zi + cr;
            im = zr * zi * 2.0 + ci;
            zr = re;
            zi = im;
            n++;
        }

        dev_a[3 * (x + y * width) + 0] = uint8_t(255 * (0.5f * sin(0.1f * n) + 0.5f));
        dev_a[3 * (x + y * width) + 1] = uint8_t(255 * (0.2f * sin(0.1f * n + 2.094f) + 0.5f));
        dev_a[3 * (x + y * width) + 2] = uint8_t(255 * (0.5f * sin(0.1f * n + 4.188f) + 0.5f));
    }
}

hipError_t calculateWithCuda(uint8_t* hst_a, const unsigned width, const unsigned height, const unsigned iterations)
{
    unsigned size = width * height * 3;
    dim3 blocks(128,72), threads(32,30);
    uint8_t *dev_a = nullptr;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffer    .
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(uint8_t));
	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffer.
    cudaStatus = hipMemcpy(dev_a, hst_a, size * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //// Get Device Properties
    hipDeviceProp_t props;
    cudaStatus = hipGetDeviceProperties(&props, 0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipGetDeviceProperties failed!");
        goto Error;
    }
    std::cout << "GPU: " << props.name << ", maxThreadsPerBlock: " << props.maxThreadsPerBlock << std::endl;

    // Launch a kernel on the GPU with one thread for each element.
    fractalKernel <<< blocks, threads >>> (dev_a, width, height, iterations);
    
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(hst_a, dev_a, size * sizeof(uint8_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_a);
    
    return cudaStatus;
}

int main()
{
    const unsigned width = 4096, height = 2160, arraySize = { width * height * 3 };
	const unsigned iterations = 1024;
	uint8_t* hst_x = new uint8_t[arraySize]{ 0 };

    // Generate Fractal using CUDA
	hipError_t cudaStatus = calculateWithCuda(hst_x, width, height, iterations);
	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "calculateWithCuda failed!");
        return 1;
	}

	// Save Fractal to a bitmap file
	Bitmap::SaveFractal(hst_x, width, height);
	
	return 0;
}