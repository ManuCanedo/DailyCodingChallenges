#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include "Bitmap.h"

#include <cstdint>
#include <math.h>
#include <string>
#include <iostream>

// Kernel executed at the Device
__global__ void fractalKernel(uint8_t* a, const unsigned width, const unsigned height, const unsigned iterations)
{
	const int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < width * height * 3)
    {
        const double xScale = (2.0) / width;
 	    const double yScale = (1.0) / height;

        double cr = (i % width) * xScale;
        double ci = (i / width) * yScale;
        double zr = 0.0;
        double zi = 0.0;

        int n = 0;

        while ((zr * zr + zi * zi) < 4.0 && n < iterations)
        {
            zr = zr * zr - zi * zi + cr;
            zi = zr * zi * 2.0 + ci;
            n++;
        }

        if (n == iterations)
            a[3 * i + 0] = uint8_t(255);
    }
}

hipError_t calculateWithCuda(uint8_t* hst_a, const unsigned width, const unsigned height, const unsigned iterations)
{
    unsigned size = width * height * 3;
    uint8_t *dev_a = nullptr;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffer    .
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(uint8_t));
	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffer.
    cudaStatus = hipMemcpy(dev_a, hst_a, size * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //// Get Device Properties
    hipDeviceProp_t props;
    cudaStatus = hipGetDeviceProperties(&props, 0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipGetDeviceProperties failed!");
        goto Error;
    }
    std::cout << "GPU: " << props.name << ", maxThreadsPerBlock: " << props.maxThreadsPerBlock << std::endl;

    // Launch a kernel on the GPU with one thread for each element.
    fractalKernel <<< std::ceil(width*height/props.maxThreadsPerBlock), props.maxThreadsPerBlock >>>(dev_a, width, height, iterations);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(hst_a, dev_a, size * sizeof(uint8_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_a);
    
    return cudaStatus;
}

int main()
{
    const unsigned width = 1280, height = 720, arraySize = { width * height * 3 };
	const unsigned iterations = 128;
	uint8_t* hst_x = new uint8_t[arraySize]{ 0 };

    // Generate Fractal using CUDA
	hipError_t cudaStatus = calculateWithCuda(hst_x, width, height, iterations);
	if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "calculateWithCuda failed!");
        return 1;
	}

	// Save Fractal to a bitmap file
	Bitmap::SaveFractal(hst_x, width, height);
	
	return 0;
}