#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include "Bitmap.h"

#include <cstdint>
#include <string>

// Function that generates a Bitmap Image with the fractal
void SaveFractal(uint8_t* pMemory, unsigned int width, unsigned int height)
{
	static uint8_t index{ 0 };

	Bitmap image(width, height);
	for (size_t y = 0; y < height; ++y)
		for (size_t x = 0; x < width; ++x)
		{
			uint8_t* pPixel = &pMemory[3 * (y * static_cast<int64_t>(width) + x)];
			image.SetPixel(x, y, pPixel[0], pPixel[1], pPixel[2]);
		}

	std::string fileName("Fractal_Screenshot_");
	fileName.append(std::to_string(index++));
	fileName.append(".bmp");

	image.Write(fileName);
}

// Function that performs the fractal computation using CUDA
__global__ void calculateFractal(uint8_t* pMemory, unsigned int width, unsigned int height, unsigned int iterations)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < width * height)
	{
		const double xScale = (3.0) / width;
		const double yScale = (2.0) / height;

		double cr = 1.0 + (i % width) * xScale;
		double ci = 2.0 + (i / width) * yScale;
		double zr = 0.0;
		double zi = 0.0;

		int n = 0;

		while ((zr * zr + zi * zi) < 4.0 && n < iterations)
		{
			zr = zr * zr - zi * zi + cr;
			zi = zr * zi * 2.0 + ci;
			n++;
		}

		if (n < iterations)
		{
			pMemory[3 * i + 0] = static_cast<uint8_t>(255);
			pMemory[3 * i + 1] = static_cast<uint8_t>(255);
			pMemory[3 * i + 2] = static_cast<uint8_t>(255);
		}
	}
}

int main()
{
	uint8_t* x;
	const unsigned int width = 1280, height = 720;
	const unsigned int N = width * height, iterations = 128;

	// Allocate Unified Memory
	hipMallocManaged(&x, N * 3 * sizeof(uint8_t));
	
	// Calculate Fractal
	calculateFractal <<<1, N>>> (x, width, height, iterations);
	hipDeviceSynchronize();

	// Save Fractal to a Bitmap
	SaveFractal(x, width, height);

	// Free Memory
	hipFree(x);
	
	return 0;
}