﻿#include "hip/hip_runtime.h"


#include <cstdint>

__global__
void calculateFractal(uint8_t* pMemory, unsigned int width, unsigned int height, unsigned int iterations)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < width * height)
	{
		const double xScale = (1.0 + 2.0) / width;
		const double yScale = (1.0 + 1.0) / height;

		double cr = 1.0 + (i % width) * xScale;
		double ci = 2.0 + (i / width) * yScale;
		double zr = 0.0;
		double zi = 0.0;

		int n = 0;

		while ((zr * zr + zi * zi) < 4.0 && n < iterations)
		{
			zr = zr * zr - zi * zi + cr;
			zi = zr * zi * 2.0 + ci;
			n++;
		}

		pMemory[3 * i + 0] = static_cast<uint8_t>(256);
		pMemory[3 * i + 1] = static_cast<uint8_t>(256);
		pMemory[3 * i + 2] = static_cast<uint8_t>(256);
	}
}

int main()
{
	uint8_t* x;
	hipMallocManaged(&x, 1280 * 720 * 3 * sizeof(uint8_t));
	
	calculateFractal <<<1, 1280 * 720>>> (x, 1280, 720, 64);
	hipDeviceSynchronize();

	hipFree(x);
	
	return 0;
}